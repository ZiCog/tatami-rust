// tatami.cu

#include <hip/hip_runtime.h>
#include <iostream>

const unsigned nMax(100000000);
const unsigned nMaxSqrt(sqrt(nMax));

__global__ void odd(unsigned* v, unsigned base)
{
    unsigned i = (blockIdx.x * blockDim.x + threadIdx.x + base) * 2 + 7;
    unsigned k2 = i + 3;
    unsigned k3 = i + i - 4;
    while ((k2 <= k3) && ((i * k2) < nMax))
    {
        unsigned k4 = (nMax - 1) / i;
        if (k3 < k4)
            k4 = k3;
        __syncthreads();
        for (unsigned j = k2 / 2; j <= k4 / 2; j++)
            atomicInc(&v[i * j], 0xffffffff);
        __syncthreads();
        k2 += i + 1;
        k3 += i - 1;
    }
}

__global__ void even(unsigned* v, unsigned base)
{
    unsigned i = (blockIdx.x * blockDim.x + threadIdx.x + base) * 2 + 8;
    unsigned k2 = i + 3;
    unsigned k3 = i + i - 4;
    while ((k2 <= k3) && ((i * k2) < nMax))
    {
        unsigned k4 = (nMax - 1) / i;
        if (k3 < k4)
            k4 = k3;
        __syncthreads();
        for (unsigned j = k2; j <= k4; ++j)
            atomicInc(&v[i * j / 2], 0xffffffff);
        __syncthreads();
        k2 += i + 1;
        k3 += i - 1;
    }
}

int Tatami(int s)
{
    unsigned* v;

    hipMalloc(&v, sizeof(unsigned) * nMax);
    hipMemset(v, 0, sizeof(unsigned) * nMax);
    const unsigned group_size = 1024;

    unsigned iterations = (nMaxSqrt - 8) / 2;
    unsigned groups = iterations / group_size;
    unsigned trailing_group_size = iterations - group_size * groups;
    even<<<groups, group_size>>>(v, 0);
    if (trailing_group_size)
        even<<<1, trailing_group_size>>>(v, groups * group_size);

    iterations = (nMaxSqrt - 7) / 2;
    groups = iterations / group_size;
    trailing_group_size = iterations - group_size * groups;
    odd<<<groups, group_size>>>(v, 0);
    if (trailing_group_size)
        odd<<<1, trailing_group_size>>>(v, groups * group_size);

    unsigned* vh = (unsigned*)malloc(sizeof(unsigned) * nMax);
    hipMemcpy(vh, v, sizeof(unsigned) * nMax, hipMemcpyDeviceToHost);

    for (unsigned i = 0; i < nMax; ++i)
        if (vh[i] == s)
            return i + i;
    return 0;  // shouldn't happen
}

int main()
{
    int s = 200;
    std::cout << "T(" << Tatami(s) << ")=" << s << std::endl;
}

