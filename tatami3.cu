#include "hip/hip_runtime.h"
// tatami.cu

#include <hip/hip_runtime.h>
#include

const unsigned nMax(100000000);
const unsigned nMaxSqrt(sqrt(nMax));

global void odd(unsigned* v, unsigned base)
{
unsigned i = (blockIdx.x * blockDim.x + threadIdx.x + base) * 2 + 7;
unsigned k2 = i + 3;
unsigned k3 = i + i - 4;
while ((k2 <= k3) && ((i * k2) < nMax))
{
unsigned k4 = (nMax - 1) / i;
if (k3 < k4)
k4 = k3;
__syncthreads();
for (unsigned j = k2 / 2; j <= k4 / 2; j++)
atomicAdd(&v[i * j], 1);
__syncthreads();
k2 += i + 1;
k3 += i - 1;
}
__syncthreads();
}

global void even(unsigned* v, unsigned base)
{
unsigned i = (blockIdx.x * blockDim.x + threadIdx.x + base) * 2 + 8;
unsigned k2 = i + 3;
unsigned k3 = i + i - 4;
while ((k2 <= k3) && ((i * k2) < nMax))
{
unsigned k4 = (nMax - 1) / i;
if (k3 < k4)
k4 = k3;
__syncthreads();
for (unsigned j = k2; j <= k4; ++j)
atomicAdd(&v[i * j / 2], 1);
__syncthreads();
k2 += i + 1;
k3 += i - 1;
}
__syncthreads();
}

int Tatami(int s)
{
unsigned* v;

hipMalloc(&v, sizeof(unsigned) * nMax);
hipMemset(v, 0, sizeof(unsigned) * nMax);
const unsigned group_size = 1024;
{
    // for (int i = 8; i < nMaxSqrt; i += 2)
    const unsigned iterations = (nMaxSqrt - 8) / 2;
    const unsigned groups = iterations / group_size;
    const unsigned trailing_group_size = iterations - group_size * groups;
    even<<<groups, group_size>>>(v, 0);
    if (trailing_group_size)
        even<<<1, trailing_group_size>>>(v, groups * group_size);
}
{
    // for (int i = 7; i < nMaxSqrt; i += 2)
    const unsigned iterations = (nMaxSqrt - 7) / 2;
    const unsigned groups = iterations / group_size;
    const unsigned trailing_group_size = iterations - group_size * groups;
    odd<<<groups, group_size>>>(v, 0);
    if (trailing_group_size)
        odd<<<1, trailing_group_size>>>(v, groups * group_size);
}

unsigned* vh = (unsigned*)malloc(sizeof(unsigned) * nMax);
hipMemcpy(vh, v, sizeof(unsigned) * nMax, hipMemcpyDeviceToHost);
hipDeviceSynchronize();

for (unsigned i = 0; i < nMax; ++i)
    if (vh[i] == s)
        return i + i;
return 0;  // shouldn't happen
}

int main()
{
int s = 200;
std::cout << "T(" << Tatami(s) << ")=" << s << std::endl;
}
